#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "warped-hashset.cu"

#define NVALS 32768
#define ELEMLEN 128
#define BLOCK 256

__global__ void kern_murmur(warped_hashset_t map, const uint32_t* vals, uint32_t n, uint32_t* out) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= n) return;
  const uint32_t* val = &vals[tid * ELEMLEN];
  out[tid] = dev_warped_hashset_insert_nonduped<murmurhash3_32>(&map, val);
}

__global__ void kern_xxhash(warped_hashset_t map, const uint32_t* vals, uint32_t n, uint32_t* out) {
  uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= n) return;
  const uint32_t* val = &vals[tid * ELEMLEN];
  out[tid] = dev_warped_hashset_insert_nonduped<xxhash32>(&map, val);
}

__global__ void kern_murmur_warped(warped_hashset_t map, const uint32_t* vals, uint32_t n, uint32_t* out) {
  uint32_t warp_id = (threadIdx.x + blockIdx.x * blockDim.x) / 32;
  uint32_t lane_id = threadIdx.x % 32;
  if (warp_id >= n/32) return;
  
  // Each thread gets its own element pointer
  uint32_t element_id = warp_id * 32 + lane_id;
  const uint32_t* val = (element_id < n) ? &vals[element_id * ELEMLEN] : nullptr;
  
  uint32_t result = dev_warped_hashset_insert_nonduped_warped<murmurhash3_32x32>(&map, val);
  
  // Store result for this thread's element
  if (element_id < n) out[element_id] = result;
}

__global__ void kern_xxhash_warped(warped_hashset_t map, const uint32_t* vals, uint32_t n, uint32_t* out) {
  uint32_t warp_id = (threadIdx.x + blockIdx.x * blockDim.x) / 32;
  uint32_t lane_id = threadIdx.x % 32;
  if (warp_id >= n/32) return;
  
  // Each thread gets its own element pointer
  uint32_t element_id = warp_id * 32 + lane_id;
  const uint32_t* val = (element_id < n) ? &vals[element_id * ELEMLEN] : nullptr;
  
  uint32_t result = dev_warped_hashset_insert_nonduped_warped<xxhash32x32>(&map, val);
  
  // Store result for this thread's element
  if (element_id < n) out[element_id] = result;
}

template <bool per_insert_print=false>
static void benchmark(const char* name, void (*kernel)(warped_hashset_t,const uint32_t*,uint32_t,uint32_t*), 
                      warped_hashset_t map, uint32_t* d_vals, uint32_t* d_out, uint32_t* h_out, int n) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  kernel<<<(n+BLOCK-1)/BLOCK,BLOCK>>>(map, d_vals, n, d_out);
  hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  hipMemcpy(h_out, d_out, sizeof(uint32_t)*n, hipMemcpyDeviceToHost);
  
  int success = 0, duplicate = 0, error = 0, uninserted = 0;
  uint64_t total_collisions = 0;
  
  printf("%s Results:\n", name);
  for (int i = 0; i < n; i++) {
    uint32_t r = h_out[i];
    if (r == NONDUPE_INSERT_ERR) {
      error++;
      if constexpr (per_insert_print) printf(" insert[%d] -> ERROR\n", i);
    } else if (r == NONDUPE_UNINSERTED) {
      uninserted++;
      if constexpr (per_insert_print) printf(" insert[%d] -> UNINSERTED (table full)\n", i);
    } else if (r == NONDUPE_INSERT_DUPE) {
      duplicate++;
      if constexpr (per_insert_print) printf(" insert[%d] -> DUPLICATE\n", i);
    } else {
      // r is the number of collisions/probes
      success++;
      total_collisions += r;
      if constexpr (per_insert_print) printf(" insert[%d] -> SUCCESS (probes: %u)\n", i, r);
    }
  }
  
  printf("Summary for %s:\n", name);
  printf(" total inserts attempted: %d\n", n);
  printf(" successful inserts: %d\n", success);
  printf(" duplicates: %d\n", duplicate);
  printf(" uninserted (table full): %d\n", uninserted);
  printf(" errors: %d\n", error);
  printf(" total collisions: %llu\n", total_collisions);
  printf(" avg collisions per success: %.2f\n", success > 0 ? (double)total_collisions / success : 0.0);
  printf(" total time: %.3f ms\n", ms);
  printf(" throughput: %.3f Mops/s\n", n/(ms*1e-3)/1e6);
  printf("---------------------------------------------------\n");
}

int main() {
  srand(time(0));
  uint32_t* h_vals = (uint32_t*)malloc(sizeof(uint32_t)*NVALS*ELEMLEN);
  for (int i = 0; i < NVALS; i++) {
    for (int j = 0; j < ELEMLEN; j++) {
      if (i == 0) h_vals[i*ELEMLEN+j] = j;
      else if (i == 1) h_vals[i*ELEMLEN+j] = j;  // Duplicate of element 0
      else if (i == 2) h_vals[i*ELEMLEN+j] = 0;
      else h_vals[i*ELEMLEN+j] = rand();
    }
  }

  uint32_t* d_vals;
  uint32_t* d_out;
  uint32_t* h_out = (uint32_t*)malloc(sizeof(uint32_t)*NVALS);
  hipMalloc(&d_vals, sizeof(uint32_t)*NVALS*ELEMLEN);
  hipMalloc(&d_out, sizeof(uint32_t)*NVALS);
  hipMemcpy(d_vals, h_vals, sizeof(uint32_t)*NVALS*ELEMLEN, hipMemcpyHostToDevice);

  warped_hashset_t map1 = warped_hashset_create<>(ELEMLEN, NVALS*2);
  benchmark("murmur", kern_murmur, map1, d_vals, d_out, h_out, NVALS);
  warped_hashset_destroy(&map1);

  warped_hashset_t map2 = warped_hashset_create<>(ELEMLEN, NVALS*2);
  benchmark("xxhash", kern_xxhash, map2, d_vals, d_out, h_out, NVALS);
  warped_hashset_destroy(&map2);

  uint32_t warped_size = ((NVALS*2+31)/32)*32;
  warped_hashset_t map3 = warped_hashset_create<true>(ELEMLEN, warped_size);
  benchmark("murmur warped", kern_murmur_warped, map3, d_vals, d_out, h_out, NVALS);
  warped_hashset_destroy(&map3);

  warped_hashset_t map4 = warped_hashset_create<true>(ELEMLEN, warped_size);
  benchmark("xxhash warped", kern_xxhash_warped, map4, d_vals, d_out, h_out, NVALS);
  warped_hashset_destroy(&map4);

  hipFree(d_vals);
  hipFree(d_out);
  free(h_vals);
  free(h_out);
  return 0;
}